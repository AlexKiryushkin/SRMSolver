
#include <gtest/gtest.h>

#include <vector>

#pragma warning(push, 0)
#include <thrust/copy.h>
#include <thrust/functional.h>
#include <thrust/logical.h>

#include "hip/hip_runtime.h"
#include ""
#pragma warning(pop)

#include <SrmSolver/gpu_matrix.h>
#include <SrmSolver/to_float.h>
#include <SrmSolver/gpu_grid.h>

#ifndef _DEBUG

namespace kae_tests {

template <class ValueT>
struct EqualToValue
{
  template <class ElemT>
  HOST_DEVICE bool operator()(ElemT value) const
  {
    return value == kae::detail::ToFloatV<ValueT, ElemT>;
  }
};

template <class T>
class gpu_matrix : public ::testing::Test {};

using TypeParams = ::testing::Types<float, double>;
TYPED_TEST_SUITE(gpu_matrix, TypeParams);

TYPED_TEST(gpu_matrix, gpu_matrix_constructor_a)
{
  using ElemType = TypeParam;
  constexpr unsigned nx{ 70U };
  constexpr unsigned ny{ 30U };
  constexpr unsigned smExtension{ 3U };
  using LxToType = std::ratio<35, 10>;
  using LyToType = std::ratio<26, 100>;
  using GpuGridType = kae::GpuGrid<nx, ny, LxToType, LyToType, smExtension, ElemType>;
  using ValueType = std::ratio<1, 1>;
  kae::GpuMatrix<GpuGridType, ElemType> matrix{ kae::detail::ToFloatV<ValueType, ElemType> };

  auto && deviceValues = matrix.values();
  auto allZeros = thrust::all_of(std::begin(deviceValues), std::end(deviceValues), EqualToValue<ValueType>{});
  EXPECT_TRUE(allZeros);
}

struct Initializer
{
  HOST_DEVICE float operator()(unsigned i, unsigned j) const
  {
    return static_cast<float>(i * i + j * j);
  }
};

TYPED_TEST(gpu_matrix, gpu_matrix_constructor_b)
{
  using ElemType = TypeParam;
  constexpr unsigned nx{ 70U };
  constexpr unsigned ny{ 30U };
  constexpr unsigned smExtension{ 3U };
  using LxToType = std::ratio<35, 10>;
  using LyToType = std::ratio<26, 100>;
  using GpuGridType = kae::GpuGrid<nx, ny, LxToType, LyToType, smExtension, ElemType>;
  kae::GpuMatrix<GpuGridType, ElemType> matrix{ Initializer{} };

  auto && deviceValues = matrix.values();
  auto matrixSize = deviceValues.size();

  std::vector<ElemType> hostValues(matrixSize);
  thrust::copy(std::begin(deviceValues), std::end(deviceValues), std::begin(hostValues));

  Initializer initializer;
  for (unsigned i = 0; i < nx; ++i)
  {
    for (unsigned j = 0; j < ny; ++j)
    {
      const auto index = j * nx + i;
      EXPECT_EQ(hostValues[index], initializer(i, j));
    }
  }
}

TYPED_TEST(gpu_matrix, gpu_matrix_values_non_const)
{
  using ElemType = TypeParam;
  constexpr unsigned nx{ 45U };
  constexpr unsigned ny{ 20U };
  constexpr unsigned smExtension{ 3U };
  using LxToType = std::ratio<35, 10>;
  using LyToType = std::ratio<26, 100>;
  using GpuGridType = kae::GpuGrid<nx, ny, LxToType, LyToType, smExtension, ElemType>;
  kae::GpuMatrix<GpuGridType, ElemType> matrix{ Initializer{} };

  auto && deviceValues = matrix.values();
  auto matrixSize = deviceValues.size();
  EXPECT_EQ(matrixSize, nx * ny);
}

TYPED_TEST(gpu_matrix, gpu_matrix_values_const)
{
  using ElemType = TypeParam;
  constexpr unsigned nx{ 45U };
  constexpr unsigned ny{ 20U };
  constexpr unsigned smExtension{ 3U };
  using LxToType = std::ratio<35, 10>;
  using LyToType = std::ratio<26, 100>;
  using GpuGridType = kae::GpuGrid<nx, ny, LxToType, LyToType, smExtension, ElemType>;
  const kae::GpuMatrix<GpuGridType, ElemType> matrix{ Initializer{} };

  const auto & deviceValues = matrix.values();
  auto matrixSize = deviceValues.size();
  EXPECT_EQ(matrixSize, nx * ny);
}

} // namespace kae_tests

#endif
