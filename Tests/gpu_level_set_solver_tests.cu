#include "hip/hip_runtime.h"

#include <gtest/gtest.h>

#include <SrmSolver/gpu_grid.h>
#include <SrmSolver/gpu_level_set_solver.h>

#include "circle_shape.h"

#ifndef _DEBUG

namespace kae_tests {

template <class T>
class gpu_level_set_solver : public ::testing::Test
{
public:

  constexpr static unsigned nx{ std::tuple_element_t<1U, T>::value };
  constexpr static unsigned ny{ std::tuple_element_t<1U, T>::value };
  constexpr static unsigned smExtension{ 3U };
  using ElemType           = std::tuple_element_t<0U, T>;
  using LxToType           = std::ratio<4, 1>;
  using LyToType           = std::ratio<4, 1>;
  using GpuGridType        = kae::GpuGrid<nx, ny, LxToType, LyToType, smExtension, ElemType>;
  using GpuMatrixType      = kae::GpuMatrix<GpuGridType, ElemType>;
  using ShapeType          = CircleShape<GpuGridType>;
  using LevelSetSolverType = kae::GpuLevelSetSolver<GpuGridType, ShapeType>;
};

using TypeParams = ::testing::Types<
  std::tuple<float,  std::integral_constant<unsigned, 100U>  >,
  std::tuple<float,  std::integral_constant<unsigned, 500U>  >,
  std::tuple<float,  std::integral_constant<unsigned, 1000U> >,
  std::tuple<double, std::integral_constant<unsigned, 100U>  >,
  std::tuple<double, std::integral_constant<unsigned, 500U>  >
>;
TYPED_TEST_SUITE(gpu_level_set_solver, TypeParams);

TYPED_TEST(gpu_level_set_solver, gpu_level_set_solver_constructor_simple)
{
  using tf              = TestFixture;
  using ElemT           = typename tf::ElemType;
  using ShapeT          = typename tf::ShapeType;
  using LevelSetSolverT = typename tf::LevelSetSolverType;

  LevelSetSolverT solver{};
  auto && deviceValues = solver.currState().values();
  const auto matrixSize = deviceValues.size();

  std::vector<ElemT> hostValues(matrixSize);
  thrust::copy(std::begin(deviceValues), std::end(deviceValues), std::begin(hostValues));

  const ShapeT shape;
  for (unsigned i = 0; i < tf::nx; ++i)
  {
    for (unsigned j = 0; j < tf::ny; ++j)
    {
      const auto index = j * tf::nx + i;
      const auto value = shape(i, j);
      const auto threshold = 10 * std::max(static_cast<ElemT>(1.0), value) * std::numeric_limits<ElemT>::epsilon();
      EXPECT_NEAR(hostValues[index], value, threshold);
    }
  }
}

TYPED_TEST(gpu_level_set_solver, gpu_level_set_solver_constructor_reinitialize)
{
  using tf              = TestFixture;
  using ElemT           = typename tf::ElemType;
  using GpuGridT        = typename tf::GpuGridType;
  using ShapeT          = typename tf::ShapeType;
  using LevelSetSolverT = typename tf::LevelSetSolverType;

  LevelSetSolverT solver{ShapeT{}, tf::nx };
  auto&& deviceValues = solver.currState().values();
  const auto matrixSize = deviceValues.size();

  std::vector<ElemT> hostValues(matrixSize);
  thrust::copy(std::begin(deviceValues), std::end(deviceValues), std::begin(hostValues));

  for (unsigned i = 0U; i < tf::nx; ++i)
  {
    for (unsigned j = 0U; j < tf::ny; ++j)
    {
      const auto index = j * tf::nx + i;
      if (std::fabs(hostValues[index]) < 10 * GpuGridT::hx)
      {
        const auto value = ShapeT::reinitializedValue(i, j);
        const auto threshold = 5 * std::max(static_cast<ElemT>(1.0), value) * GpuGridT::hx * GpuGridT::hx;
        EXPECT_NEAR(hostValues[index], value, threshold);
      }
    }
  }
}

TYPED_TEST(gpu_level_set_solver, gpu_level_set_solver_reinitialize)
{
  using tf              = TestFixture;
  using ElemT           = typename tf::ElemType;
  using GpuGridT        = typename tf::GpuGridType;
  using ShapeT          = typename tf::ShapeType;
  using LevelSetSolverT = typename tf::LevelSetSolverType;

  LevelSetSolverT solver{ ShapeT{} };
  solver.reinitialize(tf::nx);
  auto&& deviceValues = solver.currState().values();
  const auto matrixSize = deviceValues.size();

  std::vector<ElemT> hostValues(matrixSize);
  thrust::copy(std::begin(deviceValues), std::end(deviceValues), std::begin(hostValues));

  for (unsigned i = 0U; i < tf::nx; ++i)
  {
    for (unsigned j = 0U; j < tf::ny; ++j)
    {
      const auto index = j * tf::nx + i;
      if (std::fabs(hostValues[index]) < 10 * GpuGridT::hx)
      {
        const auto value = ShapeT::reinitializedValue(i, j);
        const auto threshold = 5 * std::max(static_cast<ElemT>(1.0), value) * GpuGridT::hx * GpuGridT::hx;
        EXPECT_NEAR(hostValues[index], value, threshold);
      }
    }
  }
}

TYPED_TEST(gpu_level_set_solver, gpu_level_set_solver_integrate_overload_a)
{
  using tf = TestFixture;
  using ElemT = typename tf::ElemType;
  using GpuGridT = typename tf::GpuGridType;
  using GpuMatrixT = typename tf::GpuMatrixType;
  using ShapeT = typename tf::ShapeType;
  using LevelSetSolverT = typename tf::LevelSetSolverType;

  LevelSetSolverT solver{ ShapeT{}, tf::nx };
  GpuMatrixT velocities{ static_cast<ElemT>(1.0) };
  const auto dt = solver.integrateInTime(velocities, tf::nx / 10U);

  auto&& deviceValues = solver.currState().values();
  const auto matrixSize = deviceValues.size();

  std::vector<ElemT> hostValues(matrixSize);
  thrust::copy(std::begin(deviceValues), std::end(deviceValues), std::begin(hostValues));

  for (unsigned i = 0U; i < tf::nx; ++i)
  {
    for (unsigned j = 0U; j < tf::ny; ++j)
    {
      const auto index = j * tf::nx + i;
      if (std::fabs(hostValues[index]) < 10 * GpuGridT::hx)
      {
        const auto value = ShapeT::integratedValue(i, j, dt);
        const auto threshold = 5 * std::max(static_cast<ElemT>(1.0), value) * GpuGridT::hx * GpuGridT::hx;
        EXPECT_NEAR(hostValues[index], value, threshold);
      }
    }
  }
}

TYPED_TEST(gpu_level_set_solver, gpu_level_set_solver_integrate_overload_b)
{
  using tf              = TestFixture;
  using ElemT           = typename tf::ElemType;
  using GpuGridT        = typename tf::GpuGridType;
  using GpuMatrixT      = typename tf::GpuMatrixType;
  using ShapeT          = typename tf::ShapeType;
  using LevelSetSolverT = typename tf::LevelSetSolverType;

  LevelSetSolverT solver{ ShapeT{}, tf::nx };
  GpuMatrixT velocities{ static_cast<ElemT>(1.0) };
  const auto integrateTime{ static_cast<ElemT>(0.1) };
  const auto dt = solver.integrateInTime(velocities, integrateTime);
  EXPECT_EQ(dt, integrateTime);

  auto&& deviceValues = solver.currState().values();
  const auto matrixSize = deviceValues.size();

  std::vector<ElemT> hostValues(matrixSize);
  thrust::copy(std::begin(deviceValues), std::end(deviceValues), std::begin(hostValues));

  for (unsigned i = 0U; i < tf::nx; ++i)
  {
    for (unsigned j = 0U; j < tf::ny; ++j)
    {
      const auto index = j * tf::nx + i;
      if (std::fabs(hostValues[index]) < 10 * GpuGridT::hx)
      {
        const auto value = ShapeT::integratedValue(i, j, dt);
        const auto threshold = 5 * std::max(static_cast<ElemT>(1.0), value) * GpuGridT::hx * GpuGridT::hx;
        EXPECT_NEAR(hostValues[index], value, threshold);
      }
    }
  }
}

} // namespace kae_tests

#endif
